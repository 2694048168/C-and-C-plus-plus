#include "hipError_t.cuh"
#include "add.cuh"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <iostream>

// -------------------------------------
int main(int argc, char const *argv[])
{
    const size_t arraySize = 1000000;
    const size_t bytesSize = sizeof(Precision) * arraySize;
    Precision *array_X = new Precision[arraySize]();
    Precision *array_Y = new Precision[arraySize]();
    Precision *array_Z = new Precision[arraySize]();

    for (size_t i = 0; i < arraySize; ++i)
    {
        array_X[i] = a;
        array_Y[i] = b;
    }

    /* CPU performacne
    ------------------------- */
    std::cout << "The performance of CPU\n";
    cpu_performance(array_X, array_Y, array_Z, arraySize);
    check(array_Z, arraySize);
    std::cout << "---------------------------------------\n";

    /* memory copy from HOST into DEVICE GPU.
    ------------------------------------------ */
    Precision *d_X, *d_Y, *d_Z;
    CHECK(hipMalloc(&d_X, bytesSize));
    CHECK(hipMalloc(&d_Y, bytesSize));
    CHECK(hipMalloc(&d_Z, bytesSize));

    CHECK(hipMemcpy(d_X, array_X, bytesSize, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_Y, array_Y, bytesSize, hipMemcpyHostToDevice));

    /* GPU performacne
    ------------------------- */
    std::cout << "The performance of GPU\n";
    gpu_performance(d_X, d_Y, d_Z, arraySize);
    CHECK(hipMemcpy(array_Z, d_Z, bytesSize, hipMemcpyDeviceToHost));
    check(array_Z, arraySize);
    std::cout << "---------------------------------------\n";

    delete [] array_X;
    delete [] array_Y;
    delete [] array_Z;

    CHECK(hipFree(d_X));
    CHECK(hipFree(d_Y));
    CHECK(hipFree(d_Z));

    return 0;
}
