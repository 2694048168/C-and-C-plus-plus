/**
 * @file 03_CudaErrorCheck.cu
 * @author Wei Li (weili_yzzcq@163.com)
 * @brief
 * @version 0.1
 * @date 2024-05-23
 *
 * @copyright Copyright (c) 2024
 *
 */


#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>

hipError_t ErrorCheck(hipError_t error_code, const char *filename,
                       int lineNumber) {
  if (error_code != hipSuccess) {
    printf(
        "\n[====CUDA Error]:\r\ncode=%d, name=%s, description=%s\r\nfile=%s, "
        "line%d\r\n\n",
        error_code, hipGetErrorName(error_code),
        hipGetErrorString(error_code), filename, lineNumber);
    return error_code;
  }
  return error_code;
}

__host__ void setGPU() {
  // detect the number of GPU on the computer.
  int deviceNum = 0;
  hipError_t error =
      ErrorCheck(hipGetDeviceCount(&deviceNum), __FILE__, __LINE__);

  if (error != hipSuccess || deviceNum == 0) {
    printf("No CUDA campatable GPU found!\n");
    exit(-1);
  } else {
    printf("The number of GPUs: %d\n", deviceNum);
  }

  // set the current execuate GPU.
  int deviceID = 0;
  error = ErrorCheck(hipSetDevice(deviceID), __FILE__, __LINE__);
  if (error != hipSuccess) {
    printf("Fail to set GPU-ID %d for current computing\n", deviceID);
  } else {
    printf("Set GPU-ID %d for current computing\n", deviceID);
  }
}

// device function
__device__ float add(const float x, const float y) { return x + y; }

// kernel function
__global__ void addMatrix_GPU(float *mat_A, float *mat_B, float *mat_C,
                              const int NUM) {
  const int block_id = blockIdx.x;
  const int thread_id = threadIdx.x;
  const int idx = block_id * blockDim.x + thread_id;

  // 防止并行 cuda-stream-processor 与 task-number 不能整除
  if (idx >= NUM)
    return;
  mat_C[idx] = add(mat_A[idx], mat_B[idx]);
}

// host function code
void init_data(float *addr, int elem_count) {
  for (int idx{0}; idx < elem_count; ++idx) {
    addr[idx] = (float)(rand() & 0xFF) / 10.f;
  }
}

// ------------------------------------
int main(int argc, const char **argv) {

  // ====Step1. set the GPU device
  setGPU();

  // ====Step2. malloc the memory on host, and init-memory
  const size_t element_NUM = 4096;
  const size_t byte_count = element_NUM * sizeof(float);

  float *fpHost_matA;
  float *fpHost_matB;
  float *fpHost_matC;

  fpHost_matA = (float *)malloc(byte_count);
  fpHost_matB = (float *)malloc(byte_count);
  fpHost_matC = (float *)malloc(byte_count);
  if (fpHost_matA != nullptr && fpHost_matB != nullptr &&
      fpHost_matC != nullptr) {
    memset(fpHost_matA, 0, byte_count);
    memset(fpHost_matB, 0, byte_count);
    memset(fpHost_matC, 0, byte_count);
  } else {
    printf("Fail to allocate Host memory\n");
    exit(-1);
  }

  // ====Step3. malloc the memory on device, and init-memory
  float *fpDevice_matA;
  float *fpDevice_matB;
  float *fpDevice_matC;

  hipMalloc((float **)&fpDevice_matA, byte_count);
  hipMalloc((float **)&fpDevice_matB, byte_count);
  hipMalloc((float **)&fpDevice_matC, byte_count);
  if (fpDevice_matA != nullptr && fpDevice_matB != nullptr &&
      fpDevice_matC != nullptr) {
    hipMemset(fpDevice_matA, 0, byte_count);
    hipMemset(fpDevice_matB, 0, byte_count);
    hipMemset(fpDevice_matC, 0, byte_count);
  } else {
    printf("Fail to allocate Host memory\n");

    free(fpHost_matA);
    free(fpHost_matB);
    free(fpHost_matC);

    exit(-1);
  }

  // ====Step. init the data on host
  srand(42);
  init_data(fpHost_matA, element_NUM);
  init_data(fpHost_matB, element_NUM);

  // ====Step4. Copy data from Host into Device
  hipMemcpy(fpDevice_matA, fpHost_matA, byte_count, hipMemcpyHostToDevice);
  hipMemcpy(fpDevice_matB, fpHost_matB, byte_count, hipMemcpyHostToDevice);
  hipMemcpy(fpDevice_matC, fpHost_matC, byte_count, hipMemcpyHostToDevice);

  // ====Step5. kernel function execuate on Device
  const int block_dim = 2048; //! Error: max-thread==1024
  dim3 block(block_dim);
  dim3 grid(element_NUM / block_dim);
  addMatrix_GPU<<<grid, block>>>(fpDevice_matA, fpDevice_matB, fpDevice_matC,
                                 element_NUM);

  // 捕捉核函数错误方法, 在调用核函数后，追加如下代码:
  // 第一条语句作用是捕捉第二条同步函数之前的最后一个错误;
  // 第二条语句同步主机与设备，因为CPU和GPU是异构架构
  ErrorCheck(hipGetLastError(), __FILE__, __LINE__);
  ErrorCheck(hipDeviceSynchronize(), __FILE__, __LINE__);

  //   cudaDeviceSynchronize();
  //   异构计算 Heterogeneous computing

  // ====Step6. Host get result from Device
  // cudaMemcpy function will wait kernel function over(阻塞/同步)
  hipMemcpy(fpHost_matC, fpDevice_matC, byte_count, hipMemcpyDeviceToHost);

  for (int idx{0}; idx < 10; ++idx) {
    printf("idx=%2d\tmatrix_A:%.2f\tmatrix_B:%.2f\tresult=%.2f\n", idx + 1,
           fpHost_matA[idx], fpHost_matB[idx], fpHost_matC[idx]);
  }

  // ====Step7. free and cudaFree memory
  if (fpHost_matA) {
    free(fpHost_matA);
    fpHost_matA = nullptr;
  }
  if (fpHost_matB) {
    free(fpHost_matB);
    fpHost_matB = nullptr;
  }
  if (fpHost_matC) {
    free(fpHost_matC);
    fpHost_matC = nullptr;
  }

  if (fpDevice_matA) {
    hipFree(fpDevice_matA);
    fpDevice_matA = nullptr;
  }
  if (fpDevice_matB) {
    hipFree(fpDevice_matB);
    fpDevice_matB = nullptr;
  }
  if (fpDevice_matC) {
    hipFree(fpDevice_matC);
    fpDevice_matC = nullptr;
  }

  // ====Step8. reset device
  hipDeviceReset();
  printf("Matrix-add via Heterogeneous computing successfully\n");
  return 0;
}
