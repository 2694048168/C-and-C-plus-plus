#include "hip/hip_runtime.h"
/**
 * @file 12_ReduceCompute.cu
 * @author Wei Li (weili_yzzcq@163.com)
 * @brief
 * @version 0.1
 * @date 2024-05-26
 *
 * @copyright Copyright (c) 2024
 *
 */

#include "utility.cuh"
#include <iostream>

void initialData(float *addr, int elemCount) {
  for (int i = 0; i < elemCount; i++) {
    addr[i] = (float)(rand() & 0xFF) / 10.f;
  }
  return;
}

// GPU计算数组加法
__global__ void addFromGPU(float *A, float *B, float *C, const int N) {
  const int id = threadIdx.x + blockIdx.x * blockDim.x;

  if (id >= N)
    return;
  // C[id] = add(A[id], B[id]);
  C[id] = A[id] + B[id];
}

// CPU计算数组加法
void addFromCPU(float *A, float *B, float *C, const int N) {
  for (int i = 0; i < N; i++) {
    C[i] = A[i] + B[i];
  }

  return;
}

void checkResult(float *hostRef, float *gpuRef, const int N) {
  double epsilon = 1.0E-8;
  bool match = 1;

  for (int i = 0; i < N; i++) {
    if (abs(hostRef[i] - gpuRef[i]) > epsilon) {
      match = 0;
      printf("host %f gpu %f\n", hostRef[i], gpuRef[i]);
      break;
    }
  }

  if (match)
    printf("Arrays match.\n\n");
  else
    printf("Arrays do not match.\n\n");
}

// -------------------------------------
int main(int argc, const char **argv) {
  int devID = 0;
  hipDeviceProp_t deviceProps;
  CUDA_CHECK(hipGetDeviceProperties(&deviceProps, devID));
  std::cout << "[INFO] 运行GPU设备: " << deviceProps.name << std::endl;
  std::cout << "[INFO] SM数量: " << deviceProps.multiProcessorCount
            << std::endl;
  std::cout << "[INFO] L2缓存大小: " << deviceProps.l2CacheSize / (1024 * 1024)
            << "M\n";
  std::cout << "[INFO] SM最大驻留线程数量: "
            << deviceProps.maxThreadsPerMultiProcessor << std::endl;
  std::cout << "[INFO] 设备是否支持流优先级："
            << deviceProps.streamPrioritiesSupported << std::endl;
  std::cout << "[INFO] 设备是否支持在L1缓存中缓存全局内存: "
            << deviceProps.globalL1CacheSupported << std::endl;
  std::cout << "[INFO] 设备是否支持在L1缓存中缓存本地内存: "
            << deviceProps.localL1CacheSupported << std::endl;
  std::cout << "[INFO] 一个SM可用的最大共享内存量: "
            << deviceProps.sharedMemPerMultiprocessor / 1024 << "KB\n";
  std::cout << "[INFO] 一个SM可用的32位最大寄存器数量: "
            << deviceProps.regsPerMultiprocessor / 1024 << "K\n";
  std::cout << "[INFO] 一个SM最大驻留线程块数量: "
            << deviceProps.maxBlocksPerMultiProcessor << std::endl;
  std::cout << "[INFO] GPU内存带宽: " << deviceProps.memoryBusWidth
            << std::endl;
  std::cout << "[INFO] GPU内存频率: "
            << (float)deviceProps.memoryClockRate / (1024 * 1024) << "GHz\n\n";

  // --------------------------------------------------------------
  int iElemCount = 2048;                            // 设置元素数量
  size_t stBytesCount = iElemCount * sizeof(float); // 字节数

  // 1、分配主机内存
  float *fpHost_A = nullptr;
  float *fpHost_B = nullptr;
  float *fpHost_C = nullptr;
  float *fpDeviceRef = nullptr;
  fpHost_A = (float *)malloc(stBytesCount);
  fpHost_B = (float *)malloc(stBytesCount);
  fpHost_C = (float *)malloc(stBytesCount);
  fpDeviceRef = (float *)malloc(stBytesCount);
  srand(666); // 设置随机种子
  initialData(fpHost_A, iElemCount);
  initialData(fpHost_B, iElemCount);
  memset(fpHost_C, 0, stBytesCount);
  memset(fpDeviceRef, 0, stBytesCount);

  // 2、分配设备内存
  float *fpDevice_A = nullptr;
  float *fpDevice_B = nullptr;
  float *fpDevice_C = nullptr;
  CUDA_CHECK(hipMalloc((float **)&fpDevice_A, stBytesCount));
  CUDA_CHECK(hipMalloc((float **)&fpDevice_B, stBytesCount));
  CUDA_CHECK(hipMalloc((float **)&fpDevice_C, stBytesCount));
  CUDA_CHECK(
      hipMemcpy(fpDevice_A, fpHost_A, stBytesCount, hipMemcpyHostToDevice));
  CUDA_CHECK(
      hipMemcpy(fpDevice_B, fpHost_B, stBytesCount, hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemset(fpDevice_C, 0, stBytesCount));

  // 3、CPU中进行计算
  addFromCPU(fpHost_A, fpHost_B, fpHost_C, iElemCount);
  // 4、GPU中进行计算
  dim3 block(64);
  dim3 grid((iElemCount + block.x - 1) / 64);
  addFromGPU<<<grid, block>>>(fpDevice_A, fpDevice_B, fpDevice_C,
                              iElemCount); // 调用核函数
  CUDA_CHECK(hipDeviceSynchronize());
  CUDA_CHECK(hipMemcpy(fpDeviceRef, fpDevice_C, stBytesCount,
                        hipMemcpyDeviceToHost));

  // 对比CPU与GPU计算结果
  checkResult(fpHost_C, fpDeviceRef, iElemCount);

  for (int i = 0; i < 10; i++) // 打印
  {
    printf("idx=%2d\tmatrix_A:%.2f\tmatrix_B:%.2f\tresult=%.2f\n", i + 1,
           fpHost_A[i], fpHost_B[i], fpDeviceRef[i]);
  }

  free(fpDeviceRef);
  free(fpHost_C);
  free(fpHost_B);
  free(fpHost_A);
  CUDA_CHECK(hipFree(fpDevice_C));
  CUDA_CHECK(hipFree(fpDevice_B));
  CUDA_CHECK(hipFree(fpDevice_A));

  CUDA_CHECK(hipDeviceReset());
  return 0;
}
