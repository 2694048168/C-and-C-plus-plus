#include "hip/hip_runtime.h"
#include "stdio.h"
#include<iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define NUM_THREADS 10
#define N 10

// Define texture reference for 1-d access.
// texture <> CUDA 内置类型变量。
// 具体参数列表以及使用例子，可以查看 CUDA 编程手册。
texture <float, 1, hipReadModeElementType> textureRef;

// Kernel function for using texture memory.
__global__ void gpu_texture_memory(int n, float *device_out)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < n) 
  {
		float temp = tex1D(textureRef, float(idx));
		device_out[idx] = temp;
	}
}

int main(int argc, char *argv[])
{
	// Calculate number of blocks to launch.
  int num_blocks = N / NUM_THREADS + ((N % NUM_THREADS) ? 1 : 0);
  
	// Declare device pointer.
  float *device_out;
  
	// allocate space on the device for the result.
  hipMalloc((void**)&device_out, sizeof(float) * N);
  
	// allocate space on the host for the results
  float *host_out = (float*)malloc(sizeof(float)*N);
  
	// Declare and initialize host array.
	float host_in[N];
  for (unsigned int i = 0; i < N; ++i) 
  {
		host_in[i] = float(i);
  }
  
  // Define CUDA Array.
  // CUDA 数组，hipArray CUDA 内置的数据类型。
  // 具体使用方法查看 CUDA 编程手册。
	hipArray *cu_Array;
  hipMallocArray(&cu_Array, &textureRef.channelDesc, N, 1);
  
	// Copy data to CUDA Array.
	hipMemcpyToArray(cu_Array, 0, 0, host_in, sizeof(float)*N, hipMemcpyHostToDevice);
	
	// bind a texture to the CUDA array.
	hipBindTextureToArray(textureRef, cu_Array);
	// Call Kernel.	
  gpu_texture_memory <<<num_blocks, NUM_THREADS >>> (N, device_out);
	
	// copy result back to host.
  hipMemcpy(host_out, device_out, sizeof(float)*N, hipMemcpyDeviceToHost);
  
	printf("Use of Texture memory on GPU: \n");
  for (unsigned int i = 0; i < N; ++i) 
  {
		printf("Texture element at %d is : %f\n",i, host_out[i]);
  }
  
  // Free dynamically managed memory.
	free(host_out);
	hipFree(device_out);
	hipFreeArray(cu_Array);
  hipUnbindTexture(textureRef);
  
  return 0;
}
