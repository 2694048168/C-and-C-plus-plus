#include <hip/hip_runtime.h>

#include <stdio.h>
#include <iostream>

// Define kernel function.
__global__ void gpuAdd(int *device_a, int *device_b, int *device_c)
{
  *device_c = *device_a + *device_b;
}

int main(int argc, char **argv)
{
  // Define host variables and device pointers.
  int host_a, host_b, host_c;
  int *device_a, *device_b, *device_c;
  
  // Initialize host variables.
  host_a = 11;
  host_b = 13;

  // CUDA 错误处理。
  hipError_t cudaStatus;

  // Allocate GPU buffers for three vectors(two input, ont output).
  cudaStatus = hipMalloc((void**)&device_c, sizeof(int));
  if (cudaStatus != hipSuccess)
  {
    fprintf(stderr, "hipMalloc failed.\n");
    // 跳转到标号 Error
    goto Error;
  }
  cudaStatus = hipMalloc((void**)&device_a, sizeof(int));
  if (cudaStatus != hipSuccess)
  {
    fprintf(stderr, "hipMalloc failed.\n");
    // 跳转到标号 Error
    goto Error;
  }
  cudaStatus = hipMalloc((void**)&device_b, sizeof(int));
  if (cudaStatus != hipSuccess)
  {
    fprintf(stderr, "hipMalloc failed.\n");
    // 跳转到标号 Error
    goto Error;
  }

  // Copy input vectors from host memory to GPU buffers.
  cudaStatus = hipMemcpy(device_a, &host_a, sizeof(int), hipMemcpyHostToDevice);
  if (cudaStatus != hipSuccess)
  {
    fprintf(stderr, "hipMemcpy failed.\n");
    // 跳转到标号 Error
    goto Error;
  }
  cudaStatus = hipMemcpy(device_b, &host_b, sizeof(int), hipMemcpyHostToDevice);
  if (cudaStatus != hipSuccess)
  {
    fprintf(stderr, "hipMemcpy failed.\n");
    // 跳转到标号 Error
    goto Error;
  }

  // Launch kernel on GPU device with one thread for each element.
  gpuAdd <<< 1, 1 >>> (device_a, device_b, device_c);

  // Check for any errors launching the kernel.
  cudaStatus = hipGetLastError();
  if (cudaStatus != hipSuccess)
  {
    fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
    // 跳转到标号 Error
    goto Error;
  }

  // Copy output vector from GPU device to host memory.
  cudaStatus = hipMemcpy(&host_c, device_c, sizeof(int), hipMemcpyDeviceToHost);
  if (cudaStatus != hipSuccess)
  {
    fprintf(stderr, "hipMemcpy failed.\n");
    // 跳转到标号 Error
    goto Error;
  }

  printf("Passing parameters by reference output: %d + %d = %d\n", host_a, host_b, host_c);

Error:
  // 标号处理，直接释放内存，结束程序。
  hipFree(device_a);
  hipFree(device_b);
  hipFree(device_c);

  return 0;
}